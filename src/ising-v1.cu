#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>






__global__ void ising_kernel(int *G,int *newG,double *w,int n){

  int x,y;
  double weight=0; 

  
  int id=blockIdx.x*blockDim.x+threadIdx.x;

  if(id<n*n){

    //set indexes
    int i,j;
    i=id/n; 
    j=id%n; 

    //influence of neighbors
    for(int ibor=-2;ibor<3;ibor++){
      for(int jbor=-2;jbor<3;jbor++){

         weight+=w[(ibor+2)*5+jbor+2]*G[((i-ibor+n)%n)*n +(j-jbor+n)%n ];



      }
   }
                
                
                
                
    

   
    if(weight<1e-4&&weight>-(1e-4)){
                    newG[i*n+j]=G[i*n+j];
                }else if(weight>0){
                    newG[i*n+j]=1;
                }else{
                    newG[i*n+j]=-1;

                }
  }
}






void ising( int *G, double *w, int k, int n){

  int *newG,*swapG,*G2;
  double *w2;

  hipMallocManaged(&newG,n*n*sizeof(int)); //save previous G before changing it
  hipMallocManaged(&G2,n*n*sizeof(int));
  hipMallocManaged(&w2,25*sizeof(double));
  
  hipMemcpy( w2, w,  25*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy( G2, G,  n*n*sizeof(int),hipMemcpyHostToDevice);
  
  for(int iter=0;iter<k;iter++){
	bool repeat=true;

   ising_kernel<<<n,n>>>(G2,newG,w2,n);

    
		hipDeviceSynchronize();
    

  for(int i=0;i<n;i++){
            for(int j=0;j<n;j++){
			if(repeat&&newG[i*n+j]!=G2[i*n+j]){
                      repeat=false;
            }
                 int   temp=newG[i*n+j];

                    newG[i*n+j]=G2[i*n+j];

                    G2[i*n+j]=temp;
            }
        }
  
   if(repeat){
      break;
  }
  

  }


    hipMemcpy(G, G2, n*n*sizeof(int),hipMemcpyDeviceToHost);
}








int main()
{
  printf("==========================START=============================\n");
   
    double weight[]={0.004,0.016,0.026,0.016,0.004,0.016,0.071,0.117,0.071,0.016,0.026,0.117,0,0.117,0.026,0.016,0.071,0.117,0.071,0.016,0.004,0.016,0.026,0.016,0.004};
    int n=517;
    int X[n*n];
    size_t size;

 
    FILE *fp = fopen("conf-init.bin", "rb");
    size = fread(X, sizeof(int), n * n, fp);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp);



    

    int k=1;
    

    ising(X,weight,k,n);

    int checkX[n*n];
    FILE *fp2 = fopen("conf-1.bin", "rb");
    size = fread(checkX, sizeof(int), n * n, fp2);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp2);
    bool flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX[i*n+j]!=X[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }

    if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }
	printf("k=4:\n")
	k=4	
	
	*fp = fopen("conf-init.bin", "rb");
    size = fread(X, sizeof(int), n * n, fp);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp);
	
	*fp2 = fopen("conf-4.bin", "rb");
    size = fread(checkX, sizeof(int), n * n, fp2);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp2);
    flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX[i*n+j]!=X[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }

    if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }
   
   
   
   
   
   printf("k=11:\n")
	k=11	
	
	*fp = fopen("conf-init.bin", "rb");
    size = fread(X, sizeof(int), n * n, fp);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp);
	
	*fp2 = fopen("conf-11.bin", "rb");
    size = fread(checkX, sizeof(int), n * n, fp2);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp2);
    flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX[i*n+j]!=X[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }
   
   
   
   
   printf("\n=========================END==========================");
    return 0;
}
