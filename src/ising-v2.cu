




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define BLOCK_SIZE 8
#define GRID_SIZE 8


__global__ void ising_kernel(int *G,int *newG,double *w,int n){

  
  
  int id=blockIdx.x*blockDim.x+threadIdx.x;
	
	unsigned int xBlock = blockDim.x * blockIdx.x;
	unsigned int yBlock = blockDim.y * blockIdx.y;
  
	unsigned int xIndex = xBlock + threadIdx.x;
	unsigned int yIndex = yBlock + threadIdx.y;
	
	
	unsigned int tempX = xBlock + threadIdx.x;
	unsigned int tempY = yBlock + threadIdx.y;
	
	int iterations;
	if (n%(BLOCK_SIZE*GRID_SIZE)==0){
		
		iterations=n/(BLOCK_SIZE*GRID_SIZE);
		
		
	}else{
		
		iterations=n/(BLOCK_SIZE*GRID_SIZE)+1;
		
		
	}
	
	
	
	
	
	for(int i=0;i<iterations;i++){
		xIndex=tempX+GRID_SIZE*BLOCK_SIZE*(i);
		for(int j=0;j<iterations;j++){
			yIndex=tempY+GRID_SIZE*BLOCK_SIZE*(j);
  if(xIndex<n&&yIndex<n){
	double weight=0; 
	
	

    for(int ibor=-2;ibor<3;ibor++){
      for(int jbor=-2;jbor<3;jbor++){

         weight+=w[(ibor+2)*5+jbor+2]*G[((xIndex-ibor+n)%n)*n +(yIndex-jbor+n)%n ];



      }
   }
                
                
                

    if(weight<1e-4&&weight>-(1e-4)){
                    newG[xIndex*n+yIndex]=G[xIndex*n+yIndex];
                }else if(weight>0){
                    newG[xIndex*n+yIndex]=1;
                }else{
                    newG[xIndex*n+yIndex]=-1;

                }
				
				
				

  }
  
  
  
		}
	}
}






void ising( int *G, double *w, int k, int n){

  int *newG,*G2;
  double *w2;

  hipMallocManaged(&newG,n*n*sizeof(int)); 
  hipMallocManaged(&G2,n*n*sizeof(int));
  hipMallocManaged(&w2,25*sizeof(double));
  
  hipMemcpy( w2, w,  25*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy( G2, G,  n*n*sizeof(int),hipMemcpyHostToDevice);
 // double total_time=0;
  
  
  
  for(int iter=0;iter<k;iter++){
     bool repeat=true;

     
     
  dim3 grid(GRID_SIZE, GRID_SIZE);
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
 
//  struct timeval startwtime, endwtime;
//      gettimeofday (&startwtime, NULL);
   ising_kernel<<<grid,block>>>(G2,newG,w2,n);
		hipDeviceSynchronize();
 //gettimeofday (&endwtime, NULL);
   //double time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+ endwtime.tv_sec - startwtime.tv_sec);
    //        total_time+=time;
    

  for(int i=0;i<n;i++){
            for(int j=0;j<n;j++){
                if(repeat&&newG[i*n+j]!=G2[i*n+j]){
                      repeat=false;
            }
                 int   temp=newG[i*n+j];

                    newG[i*n+j]=G2[i*n+j];

                    G2[i*n+j]=temp;
            }
        }
  
      if(repeat){
          break;
      }
  

  }


 
    hipMemcpy(G, G2, n*n*sizeof(int),hipMemcpyDeviceToHost);
 //   printf("Seconds are %lf",total_time);
}






int main()
{
  printf("=================START=========================\n");
   
    double weight[]={0.004,0.016,0.026,0.016,0.004,0.016,0.071,0.117,0.071,0.016,0.026,0.117,0,0.117,0.026,0.016,0.071,0.117,0.071,0.016,0.004,0.016,0.026,0.016,0.004};
    int n=517;
    int X[n*n];
    size_t size;

 
    FILE *fp = fopen("conf-init.bin", "rb");
    size = fread(X, sizeof(int), n * n, fp);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp);



    

    int k=11;
    

    ising(X,weight,k,n);

    int checkX[n*n];
    FILE *fp2 = fopen("conf-11.bin", "rb");
    size = fread(checkX, sizeof(int), n * n, fp2);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp2);
    bool flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX[i*n+j]!=X[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }

    if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }

    printf("\n================END==============\n");
    return 0;
}