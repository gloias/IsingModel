
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#define BLOCK_SIZE 8
#define GRID_SIZE 8



//struct timespec start, finish;
//double elapsed;


__global__ void ising_kernel(int *G,int *newG,double *w,int n){

  int x,y; 
  
	__shared__ double shared_w[25];
	__shared__ int shared_G[(BLOCK_SIZE+4)*(BLOCK_SIZE+4)];
	for(int i=0;i<25;i++){
		shared_w[i]=w[i];
	}
	
 
	
	unsigned int xBlock = blockDim.x * blockIdx.x;
	unsigned int yBlock = blockDim.y * blockIdx.y;
  
	unsigned int xIndex = xBlock + threadIdx.x;
	unsigned int yIndex = yBlock + threadIdx.y;
	
	
	unsigned int tempX = xBlock + threadIdx.x;
	unsigned int tempY = yBlock + threadIdx.y;
	
	int iterations;
	if (n%(BLOCK_SIZE*GRID_SIZE)==0){
		
		iterations=n/(BLOCK_SIZE*GRID_SIZE);
		
		
	}else{
		
		iterations=n/(BLOCK_SIZE*GRID_SIZE)+1;
		
		
	}
	
	
	
	
	
	for(int i=0;i<iterations;i++){
		xIndex=tempX+GRID_SIZE*BLOCK_SIZE*(i);
		for(int j=0;j<iterations;j++){
			yIndex=tempY+GRID_SIZE*BLOCK_SIZE*(j);
			
			
			shared_G[(threadIdx.x+2)*(BLOCK_SIZE+4)+threadIdx.y+2]=G[((xIndex+n)%n)*n+(yIndex+n)%n];
			
			if(threadIdx.x==0){
				if (threadIdx.y==0){
					for(int k=0;k<3;k++){
						for(int l=0;l<3;l++){
							if(!(k==0&&l==0)){
						shared_G[(2-k)*(BLOCK_SIZE+4)+(2-l)]=G[((xIndex-k+n)%n)*n+(yIndex-l+n)%n];
							}
						}
						
					}
						
					}else if(threadIdx.y==BLOCK_SIZE-1){
						for(int k=0;k<3;k++){
							for(int l=0;l<3;l++){
								if(!(k==0&&l==0)){
						shared_G[(2-k)*(BLOCK_SIZE+4)+(2+l+threadIdx.y)]=G[((xIndex-k+n)%n)*n+(yIndex+l+n)%n];
								}
							}
					}
						
					}else{ 
						shared_G[(2-1)*(BLOCK_SIZE+4)+(2+threadIdx.y)]=G[((xIndex-1+n)%n)*n+(yIndex+n)%n];
						shared_G[(2-2)*(BLOCK_SIZE+4)+(2+threadIdx.y)]=G[((xIndex-2+n)%n)*n+(yIndex+n)%n];
						
					}
					
					
					
				}else if(threadIdx.x==BLOCK_SIZE-1){
					if (threadIdx.y==0){
					for(int k=0;k<3;k++){
							for(int l=0;l<3;l++){
								if(!(k==0&&l==0)){
						shared_G[(2+k+threadIdx.x)*(BLOCK_SIZE+4)+(2-l)]=G[((xIndex+k+n)%n)*n+(yIndex-l+n)%n];
								}
							}
					}
					}else if(threadIdx.y==BLOCK_SIZE-1){
						
						for(int k=0;k<3;k++){
							for(int l=0;l<3;l++){
								if(!(k==0&&l==0)){
						shared_G[(2+k+threadIdx.x)*(BLOCK_SIZE+4)+(2+l+threadIdx.y)]=G[((xIndex+k+n)%n)*n+(yIndex+l+n)%n];
								}
							}
						}
						
					}else {
						
						shared_G[(2+1+threadIdx.x)*(BLOCK_SIZE+4)+(2+threadIdx.y)]=G[((xIndex+1+n)%n)*n+(yIndex+n)%n];
						shared_G[(2+2+threadIdx.x)*(BLOCK_SIZE+4)+(2+threadIdx.y)]=G[((xIndex+2+n)%n)*n+(yIndex+n)%n];
					}
					
					
				}else{
					if(threadIdx.y==0){ 
						
						shared_G[(2+threadIdx.x)*(BLOCK_SIZE+4)+(2-1)]=G[((xIndex+n)%n)*n+(yIndex-1+n)%n];
						shared_G[(2+threadIdx.x)*(BLOCK_SIZE+4)+(2-2)]=G[((xIndex+n)%n)*n+(yIndex-2+n)%n];
						
					}else if(threadIdx.y==BLOCK_SIZE-1){ 
						shared_G[(2+threadIdx.x)*(BLOCK_SIZE+4)+(2+1+threadIdx.y)]=G[((xIndex+n)%n)*n+(yIndex+1+n)%n];
						shared_G[(2+threadIdx.x)*(BLOCK_SIZE+4)+(2+2+threadIdx.y)]=G[((xIndex+n)%n)*n+(yIndex+2+n)%n];
						
					}
					
					
				}
					__syncthreads();
			
			
		
			
			
			
			
			
  if(xIndex<n&&yIndex<n){
	double weight=0; 
	
	//printf("BLOCK IDX X: %d\n",blockIdx.x);
	//printf("BLOCK IDX Y: %d\n",blockIdx.y);
	

    
    
    
	
    for(int ibor=-2;ibor<3;ibor++){
      for(int jbor=-2;jbor<3;jbor++){

		 weight+=shared_w[(ibor+2)*5+jbor+2]*shared_G[(threadIdx.x+2+ibor)*(BLOCK_SIZE+4) +(threadIdx.y+2+jbor) ];


      }
   }
   
   
  

    
    if(weight<1e-4&&weight>-(1e-4)){
                   // newG[xIndex*n+yIndex]=G[xIndex*n+yIndex];
                    newG[xIndex*n+yIndex]=shared_G[(threadIdx.x+2)*(BLOCK_SIZE+4)+threadIdx.y+2];
                }else if(weight>0){
                    newG[xIndex*n+yIndex]=1;
                }else{
                    newG[xIndex*n+yIndex]=-1;

                }
				
				
				

  }
  
  
  }
  
  
  
		}
	}








void ising( int *G, double *w, int k, int n){

  int *newG,*swapG,*G2;
  double *w2;

  hipMallocManaged(&newG,n*n*sizeof(int)); 
  hipMallocManaged(&G2,n*n*sizeof(int));
  hipMallocManaged(&w2,25*sizeof(double));
  
  hipMemcpy( w2, w,  25*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy( G2, G,  n*n*sizeof(int),hipMemcpyHostToDevice);
  double total_time=0;
  
  
  
  for(int iter=0;iter<k;iter++){
     
     int grid_dimension;
     bool repeat=true;
     
     
  dim3 grid(GRID_SIZE, GRID_SIZE);
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
// struct timeval startwtime, endwtime;
//       gettimeofday (&startwtime, NULL);
   ising_kernel<<<grid,block>>>(G2,newG,w2,n);
		hipDeviceSynchronize();
// gettimeofday (&endwtime, NULL);
//   double time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6   + endwtime.tv_sec - startwtime.tv_sec);
 //           total_time+=time;
  for(int i=0;i<n;i++){
            for(int j=0;j<n;j++){
                
                if(repeat&&newG[i*n+j]!=G2[i*n+j]){
                      repeat=false;
            }
                 int   temp=newG[i*n+j];

                    newG[i*n+j]=G2[i*n+j];

                    G2[i*n+j]=temp;
            }
        }
        
  if(repeat){
      break;
  }
 
  

  }

 
    hipMemcpy(G, G2, n*n*sizeof(int),hipMemcpyDeviceToHost);
   // printf("Seconds are %lf ",total_time);
}


int main()
{
  printf("==========================START=============================\n");
   
    double weight[]={0.004,0.016,0.026,0.016,0.004,0.016,0.071,0.117,0.071,0.016,0.026,0.117,0,0.117,0.026,0.016,0.071,0.117,0.071,0.016,0.004,0.016,0.026,0.016,0.004};
    int n=517;
    int X[n*n];
    size_t size;

 
    FILE *fp = fopen("conf-init.bin", "rb");
    size = fread(X, sizeof(int), n * n, fp);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp);



    

    int k=1;
    

    ising(X,weight,k,n);

    int checkX[n*n];
    printf("k=1:\n");
    FILE *fp2 = fopen("conf-1.bin", "rb");
    size = fread(checkX, sizeof(int), n * n, fp2);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp2);
    bool flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX[i*n+j]!=X[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }

    if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }
 


  
	printf("k=4:\n");
	k=4	;
	int X2[n*n];

	FILE *fpA = fopen("conf-init.bin", "rb");
    size = fread(X2, sizeof(int), n * n, fpA);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fpA);
	int checkX2[n*n];
	FILE *fp3 = fopen("conf-4.bin", "rb");
    size = fread(checkX2, sizeof(int), n * n, fp3);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp3);
 
 ising(X2,weight,k,n);
    flag=true;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX2[i*n+j]!=X2[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }

    if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }
   
   
   
   
   
   printf("k=11:\n");
	k=11	;
	int X3[n*n];
	FILE *fpB = fopen("conf-init.bin", "rb");
    size = fread(X3, sizeof(int), n * n, fpB);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fpB);
	int checkX3[n*n];
	FILE *fp4 = fopen("conf-11.bin", "rb");
    size = fread(checkX3, sizeof(int), n * n, fp4);
    if(size!=n*n) exit(EXIT_FAILURE);
    fclose(fp4);
    flag=true;
ising(X3,weight,k,n);
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if(checkX3[i*n+j]!=X3[i*n+j]){
                printf("\nWRONG IMPLEMENTATION\n");
                flag=false;
                break;
            }


        }
        if(!flag){
            break;
        }
    }
   
  if(flag){
        printf("\nCORRECT IMPLEMENTATION\n");
    }
      
   
   
   printf("\n=========================END==========================");
    return 0;
}